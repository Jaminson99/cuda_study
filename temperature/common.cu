#include "common.cuh"


double get_time()
{
	LARGE_INTEGER timer;
	static LARGE_INTEGER fre;
	static int init = 0;
	double t;

	if (init != 1) {
		QueryPerformanceFrequency(&fre);
		init = 1;
	}

	QueryPerformanceCounter(&timer);

	t = timer.QuadPart * 1. / fre.QuadPart;

	return t;
}


hipError_t Error_Check(hipError_t error_code, const char* file, int line)
{
	if (error_code != hipSuccess)
	{
		printf("CUDA error:\r\ncode=%d, name=%s, description=%s\r\nfile=%s, line=%d\r\n",
			error_code, hipGetErrorName(error_code),
			hipGetErrorString(error_code), file, line
		);
		exit(-1);
	}
	return error_code;
}
#define ErrorCheck( err ) (Error_Check( err, __FILE__, __LINE__ ))


void setGPU()
{
	int deviceNum = 0;
	hipError_t error = ErrorCheck(hipGetDeviceCount(&deviceNum));

	if (error != hipSuccess || deviceNum == 0)
	{
		printf("None CUDA compatible GPU found\n");
		exit(-1);
	}
	else
	{
		printf("The num of GPU is %d.\n", deviceNum);
	}

	int device = 0;
	error = ErrorCheck(hipSetDevice(device));
	if (error != hipSuccess)
	{
		printf("Fail to set GPU 0 for computing\n");
		exit(-1);
	}
	else
	{
		printf("Set GPU 0\n");
	}
}
