#include "hip/hip_runtime.h"
#include "common.cuh"


__global__ void vectorSum(float* vectorA, float* vectorB, float* result, int length)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int id = tid + bid * blockDim.x;

	if (id < length)
	{
		result[id] = vectorA[id] + vectorB[id];
	}

}



int main(void)
{
	setGPU();
	int nLength = 1 << 15;
	int nBytes = nLength * sizeof(float);

	float* hostA, * hostB, * hostResult;
	hipHostAlloc((void**)&hostA, nBytes, hipHostMallocDefault);
	hipHostAlloc((void**)&hostB, nBytes, hipHostMallocDefault);
	hipHostAlloc((void**)&hostResult, nBytes, hipHostMallocDefault);
	for (int i = 0; i < nLength; ++i)
	{
		hostA[i] = nLength - i;
		hostB[i] = i;
	}

	float* deviceA, * deviceB, * deviceResult;
	hipMalloc((void**)&deviceA, nBytes);
	hipMalloc((void**)&deviceB, nBytes);
	hipMalloc((void**)&deviceResult, nBytes);
	hipMemcpy(deviceA, hostA, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, nBytes, hipMemcpyHostToDevice);

	int blockSize = 256;
	int gridSize = nLength / blockSize;

	// create streams
	int n_stream = 4;
	hipStream_t* streams = (hipStream_t*)malloc(n_stream * sizeof(hipStream_t));
	for (int i = 0; i < n_stream; ++i)
	{
		hipStreamCreate(&streams[i]);
	}

	
	// warmup
	vectorSum <<<gridSize, blockSize>>> (deviceA, deviceB, deviceResult, nLength);

	// syn
	double beginTime = get_time();
	hipMemcpy(deviceA, hostA, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, nBytes, hipMemcpyHostToDevice);
	vectorSum <<<gridSize, blockSize>>> (deviceA, deviceB, deviceResult, nLength);
	hipMemcpy(hostResult, deviceResult, nBytes, hipMemcpyDeviceToHost);
	double endTime = get_time();
	printf("\noriginal time: %.8f", endTime - beginTime);


	// asyn
	int iLength = nLength / n_stream;
	int iBytes = nBytes / n_stream;
	beginTime = get_time();
	for (int i = 0; i < n_stream; ++i)
	{
		int ioffset = iLength * i;
		hipMemcpyAsync(&deviceA[ioffset], &hostA[ioffset], iBytes,
			hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(&deviceB[ioffset], &hostB[ioffset], iBytes,
			hipMemcpyHostToDevice, streams[i]);
		vectorSum <<<gridSize/n_stream, blockSize, 0, streams[i]>>> (&deviceA[ioffset], &deviceB[ioffset], &deviceResult[ioffset], iLength);
		hipMemcpyAsync(&hostResult[ioffset], &deviceResult[ioffset], iBytes,
			hipMemcpyDeviceToHost, streams[i]);
	}
	endTime = get_time();
	printf("\nasyn time: %.8f", endTime - beginTime);

	printf("\n");
	for (int i = 0; i < 10; ++i) printf("%.1f ", hostResult[i]);

	hipHostFree(hostA);
	hipHostFree(hostB);
	hipHostFree(hostResult);

	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceResult);
}

